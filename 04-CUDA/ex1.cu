#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

using namespace std;

__constant__ float filter[3] = {0.2,0.6,0.2};

__global__ void apply_filter(float *x, float *y){
    // Your code goes here
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index>0 && index<1023)
      y[index] += filter[0]*x[index-1] + filter[1]*x[index] + filter[2]*x[index+1];  
    // till here
}

int main(){
    float X_hs[N], *X_dev;
    float Y_hs[N], *Y_dev;

    // Initialise vector
    for(int i=0;i<N;i++){
        X_hs[i] = i * 1.0;
        Y_hs[i] = 0.0;
    }
    // Allocate memory in GPU (device)
    hipMalloc(&X_dev, N * sizeof(float));
    hipMalloc(&Y_dev, N * sizeof(float));

    // Transfer data to GPU
    hipMemcpy(X_dev, X_hs, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Y_dev, Y_hs, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch Kernel
     apply_filter<<<32,32>>>(X_dev, Y_dev);

    // Transfer data back to CPU
    hipMemcpy(Y_hs, Y_dev, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory in GPU
    for(int i=0;i<10;i++){
        cout << Y_hs[i] << endl;
    }
    hipFree(X_hs);
    hipFree(Y_hs);

    return 0;
}